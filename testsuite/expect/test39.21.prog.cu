// Slurm regression test39.21.prog.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1024;
	int i;
	float *x, *y;
	float maxError = 0.0f;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 256 elements at a time on the GPU
	add<<<1, 256>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	for (i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
